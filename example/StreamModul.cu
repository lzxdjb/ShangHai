#include "src/consolidate.cuh"
using namespace std;
#include <unistd.h>

void DebugQ( uint8_t * query)
{
    for(int b = 0 ; b < bs ; b++)
    {
        for(int n = 0 ; n < nh ; n ++)
        {
            std::cout<<endl;
            for(int q = 0 ; q < qlen ; q ++)
            {
                for(int d = 0 ; d < dim ; d++)
                {
                    std::cout<<static_cast<int>(query[b * nh * qlen * dim + n * qlen * dim + q * dim + d])<<" ";
                    // std::cout<<static_cast<int>(uint8_t(1));
                }
                std::cout<<endl;
            }
        }
    }
}

void DebugK( uint8_t * query)
{
    for(int b = 0 ; b < bs ; b++)
    {
        for(int n = 0 ; n < nh ; n ++)
        {
            std::cout<<endl;
            for(int k = 0 ; k < klen ; k ++)
            {
                for(int d = 0 ; d < dim ; d++)
                {
                    std::cout<<static_cast<int>(query[b * nh * klen * dim + n * klen * dim + k * dim + d])<<" ";
                    // std::cout<<static_cast<int>(uint8_t(1));
                }
                std::cout<<endl;
            }
        }
    }
}

void DebugLUT(float * LUT)
{
    for(int n = 0 ; n < nh ; n++)
    {
        std::cout<<endl;
        for(int k1 = 0 ; k1 < K ; k1 ++)
        {
            for(int k2 = 0 ; k2 < K ; k2++)
            {
               std::cout<<static_cast<int>(LUT[n * K * K +  k1 * K + k2])<<" ";
            }
            std::cout<<endl;
        }
    }
}

void DebugStore(float * store)
{
     for(int b = 0 ; b < bs ; b++)
    {
        for(int n = 0 ; n < nh ; n ++)
        {
            std::cout<<endl;
            for(int q = 0 ; q < qlen ; q ++)
            {
                for(int k = 0 ; k < klen ; k++)
                {
                    std::cout<<static_cast<int>(store[b * nh * qlen * klen + n * qlen * klen + q * klen + k])<<" ";
                }
            }
        }
    }
}


void Initial( uint8_t * query , uint8_t * key , float * LUT , float * store)
{
    for(int b = 0 ; b < bs ; b++)
    {
        for(int n = 0 ; n < nh ; n ++)
        {
            for(int q = 0 ; q < qlen ; q ++)
            {
                for(int d = 0 ; d < dim ; d++)
                {
                    query[b * nh * qlen * dim + n * qlen * dim + q * dim + d] = 1;
                }
            }
        }
    }

    for(int b = 0 ; b < bs ; b++)
    {
        for(int n = 0 ; n < nh ; n ++)
        {
            for(int k = 0 ; k < klen ; k ++)
            {
                for(int d = 0 ; d < dim ; d++)
                {
                    key[b * nh * klen * dim + n * klen * dim + k * dim + d] = 2;
                }
            }
        }
    }


    for(int n = 0 ; n < nh ; n++)
    {
        for(int k1 = 0 ; k1 < K ; k1 ++)
        {
            for(int k2 = 0 ; k2 < K ; k2++)
            {
                LUT[n * K * K +  k1 * K + k2] = 3;
            }
        }
    }

//////////// No need to do
    for(int b = 0 ; b < bs ; b++)
    {
        for(int n = 0 ; n < nh ; n ++)
        {
            for(int q = 0 ; q < qlen ; q ++)
            {
                for(int k = 0 ; k < klen ; k++)
                {
                    store[b * nh * qlen * klen + n * qlen * klen + q * klen + k] = 10;
                }
            }
        }
    }


}


int main() {
    
    uint8_t * query = (uint8_t *)malloc(sizeof(uint8_t) * bs * nh * qlen * dim);

    uint8_t * key = (uint8_t *)malloc(sizeof(uint8_t) * bs * nh * klen * dim);

    float * LUT = (float *)malloc(sizeof(float) * nh * K * K);

    // float * store = (float *)malloc(sizeof(float) * bs * nh * qlen * klen);

    float * store;
    checkCudaErrors( hipHostMalloc((void**)&store, sizeof(float) * bs * nh * qlen * klen) ) ;   


    Initial(query , key , LUT , store);
    // DebugQ(query);
    // DebugK(key);
    // exit(0);
    // DebugLUT(LUT);
    // DebugLUT(LUT);
    // DebugStore(store);

    // int thread = 1024;
    // if (qlen * klen < thread)
    // {
    //     thread = 1;
    //     while(1)
    //     {
    //         if (thread > qlen * klen)
    //         {

    //         }
    //     }
    // }

    int count = 1 ; 
    for(int i = 0 ; i < 3 ; i ++)
    {
        solve_stream_kernel(query , key , LUT , store);

        hipDeviceSynchronize();
        // sleep(1);

    }

 
}